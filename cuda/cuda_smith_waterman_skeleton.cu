#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <chrono>

using namespace std;

#include "cuda_smith_waterman.h"

/*
 *  You can add helper functions and variables as you wish.
 */


#define max(a, b) (a > b ? a : b)
#define min(a, b) (a < b ? a : b)

inline __device__ int idx(int i, int j, int I) {
  return j * I + i;
}

inline __device__ int get(int* scores, int i, int j, int I) {
  int value;
  while((value = scores[idx(i,j,I)]) == -1);
  return value;
}

__global__ void cuda_sw(char *a, char *b, int a_len, int b_len, int *score, int len, int* max_scores) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int tn = blockDim.x * gridDim.x;
  int prevy = 0;
  int I = b_len + 1;
  int J = a_len + 1;
  int max_score = 0;
  for (int id = tid; id < len; id += tn) {
    int x = id % I;
    int y = id / I;
//    if (y > prevy) {
//      __syncthreads();
//      prevy++;
//    }
//    if(prevy >= a_len+b_len+1){
//      break;
//    }
//    if(id >= len){
//      continue;
//    }
    int j = y - x;
    int i = x;
    if (i >= 1 && j >= 1 && i < I && j < J) {
      int s = 0;
      if(x>=1 && y >=2){
        int ad = sub_mat(a[i - 1], b[j - 1]);
        s = max(0, get(score,x - 1, y - 2, b_len + 1) + ad); //
      }
      s = max(s, get(score,x - 1, y - 1, b_len + 1) - GAP);
      s = max(s, get(score,x, y - 1, b_len + 1) - GAP);
      max_score = max(max_score, s);
      score[id] = s;
    } else {
      score[id] = 0;
    }

  }
  max_scores[tid] = max_score;
}

__global__ void Kernel() {}

int smith_waterman(int blocks_per_grid, int threads_per_block, char *_a, char *_b, int _a_len, int _b_len) {
  dim3 blocks(blocks_per_grid);
  dim3 threads(threads_per_block);

  char *a;
  char *b;
  int a_len;
  int b_len;
  int len;
  int *d_scores;
  int max_score = 0;
  int tn = blocks_per_grid * threads_per_block;
  int *d_max_scores;
  char *d_a, *d_b;

  a = _a, b = _b, a_len = _a_len, b_len = _b_len;
  if (b_len < a_len) {
    a_len = _b_len;
    b_len = _a_len;
    a = _b;
    b = _a;
  }
  len = ((b_len + 1) * (a_len + 1 + b_len));
//  scores = new int[(sizeof(int) * len)];
  hipMalloc(&d_scores, sizeof(int) * len);
  hipMalloc(&d_max_scores, sizeof(int) * tn);
  hipMalloc(&d_a, sizeof(int) * a_len);
  hipMemcpy(d_a, a, sizeof(int) * a_len, hipMemcpyHostToDevice);
  hipMalloc(&d_b, sizeof(int) * b_len);
  hipMemcpy(d_b, b, sizeof(int) * b_len, hipMemcpyHostToDevice);
  hipMemset(d_scores, 0, sizeof(int) * len);
  hipMemset(d_max_scores, -1, sizeof(int) * tn);

//  cuda_sw<<<blocks, threads>>>(a, b, a_len, b_len, d_scores, len);
  cuda_sw << < blocks, threads >> > (d_a, d_b, a_len, b_len, d_scores, len, d_max_scores);

  int *h_max = new int[tn];
  int *scores = new int[len];
  hipMemcpy(h_max, d_max_scores, sizeof(int)*tn, hipMemcpyDeviceToHost);
  hipMemcpy(scores, d_scores, sizeof(int)*len, hipMemcpyDeviceToHost);

  for(int id = 0; id < len; id++){
    if(id % (b_len+1) == 0) std::cout<<std::endl;
    std::cout<<scores[id]<<" ";
  }
  std::cout<<endl;

  for(int i = 0; i< tn; i++){
    max_score = max(max_score, h_max[i]);
  }

//  hipMemcpy(&max_score, &d_scores[len - 1], sizeof(int), hipMemcpyDeviceToHost);
  std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
  return max_score;
}
