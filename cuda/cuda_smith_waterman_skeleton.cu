#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <chrono>

using namespace std;

#include "cuda_smith_waterman.h"

/*
 *  You can add helper functions and variables as you wish.
 */

#define max(a, b) (a > b ? a : b)
#define min(a, b) (a < b ? a : b)

inline __device__ int idx(int i, int j, int I) {
  return j * I + i;
}

__global__ void cuda_sw(char *a, char *b, int a_len, int b_len, int *score, int *max_scores, int y) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int tn = blockDim.x * gridDim.x;
  int max_score = 0;
  for (int x = tid; x < b_len + 1; x += tn) {
    int j = y - x + 1;
    int i = x;
    int s = 0;
    if (i >= 1 && j >= 1 && i <= b_len && j <= a_len) {
      if (y >= 2) {
        int ad = sub_mat(a[j - 1], b[i - 1]);
        s = max(0, score[idx(x - 1, y - 2, b_len + 1)] + ad);
      }
      s = max(s, score[idx(x - 1, y - 1, b_len + 1)] - GAP);
      s = max(s, score[idx(x, y - 1, b_len + 1)] - GAP);
      max_score = max(max_score, s);
    }
    score[idx(x, y, b_len + 1)] = s;
  }
  max_scores[tid] = max(max_scores[tid], max_score);
}

__global__ void maxOf(int *max_scores) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int tn = blockDim.x * gridDim.x;
  int l = tn;
  while (l > 1) {
    __syncthreads();
    l = (l + 1) / 2;
    if (tid < l && tid + l < tn) {
      max_scores[tid] = max(max_scores[tid], max_scores[tid + l]);
    }
  }
}

int smith_waterman(int blocks_per_grid, int threads_per_block, char *_a, char *_b, int _a_len, int _b_len) {
  dim3 blocks(blocks_per_grid);
  dim3 threads(threads_per_block);

  char *a;
  char *b;
  int a_len;
  int b_len;
  int len;
  int *d_scores;
  int max_score = 0;
  int tn = blocks_per_grid * threads_per_block;
  int *d_max_scores;
  char *d_a, *d_b;

  a = _a, b = _b, a_len = _a_len, b_len = _b_len;
  if (b_len < a_len) {
    a_len = _b_len;
    b_len = _a_len;
    a = _b;
    b = _a;
  }
  len = ((b_len + 1) * (a_len + 1 + b_len));
//  scores = new int[(sizeof(int) * len)];
  hipMalloc(&d_scores, sizeof(int) * len);
  hipMalloc(&d_max_scores, sizeof(int) * tn);
  hipMalloc(&d_a, sizeof(int) * a_len);
  hipMemcpy(d_a, a, sizeof(int) * a_len, hipMemcpyHostToDevice);
  hipMalloc(&d_b, sizeof(int) * b_len);
  hipMemcpy(d_b, b, sizeof(int) * b_len, hipMemcpyHostToDevice);
  hipMemset(d_scores, 0, sizeof(int) * len);
  hipMemset(d_max_scores, 0, sizeof(int) * tn);

//  cuda_sw<<<blocks, threads>>>(a, b, a_len, b_len, d_scores, len);
  for (int y = 0; y < a_len + b_len + 1; y++) {
    cuda_sw <<< blocks, threads >>>(d_a, d_b, a_len, b_len, d_scores, d_max_scores, y);
  }
  maxOf<<< blocks, threads >>>(d_max_scores);

  int *h_max_scores = new int[tn];
//  int *scores = new int[len];
  hipMemcpy(h_max_scores, d_max_scores, sizeof(int) * tn, hipMemcpyDeviceToHost);

//  hipMemcpy(scores, d_scores, sizeof(int) * len, hipMemcpyDeviceToHost);
//  for (int id = 0; id < len; id++) {
//    if (id % (b_len + 1) == 0) std::cout << std::endl;
//    std::cout << scores[id] << " ";
//  }
//  std::cout << endl;

  for (int i = 0; i < tn; i++) {
//    std::cout << h_max_scores[i] << " ";
    max_score = max(max_score, h_max_scores[i]);
  }

//  hipMemcpy(&max_score, &d_scores[len - 1], sizeof(int), hipMemcpyDeviceToHost);
//  std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
  return max_score;
}
