#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <algorithm>
#include <chrono>

using namespace std;

#include "cuda_smith_waterman.h"

/*
 *  You can add helper functions and variables as you wish.
 */

#define max(a, b) (a > b ? a : b)

__global__ void cuda_sw(char *a, char *b, int a_len, int b_len, int *d_scores, int *max_scores, int y, int* d_scores1, int* d_scores2) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  int tn = blockDim.x * gridDim.x;
  int max_score = 0;
  for (int x = tid; x < b_len + 1; x += tn) {
    int j = y - x + 1;
    int i = x;
    int s = 0;
    if (i >= 1 && j >= 1 && i <= b_len && j <= a_len) {
      if (y >= 2) {
        int ad = sub_mat(a[j - 1], b[i - 1]);
        s = max(0, d_scores2[x-1] + ad);
      }
      s = max(s, d_scores1[x-1] - GAP);
      s = max(s, d_scores1[x] - GAP);
      max_score = max(max_score, s);
    }
    d_scores[x] = s;
  }
  max_scores[tid] = max(max_scores[tid], max_score);
}

int smith_waterman(int blocks_per_grid, int threads_per_block, char *_a, char *_b, int _a_len, int _b_len) {
  dim3 blocks(blocks_per_grid);
  dim3 threads(threads_per_block);

  char *a;
  char *b;
  int a_len;
  int b_len;
  int *d_scores, *d_scores1, *d_scores2;
  int max_score = 0;
  int tn = blocks_per_grid * threads_per_block;
  int *d_max_scores;
  char *d_a, *d_b;

  a = _a, b = _b, a_len = _a_len, b_len = _b_len;
  if (b_len < a_len) {
    a_len = _b_len;
    b_len = _a_len;
    a = _b;
    b = _a;
  }
  hipMalloc(&d_scores, sizeof(int) * (b_len + 1));
  hipMalloc(&d_scores1, sizeof(int) * (b_len + 1));
  hipMalloc(&d_scores2, sizeof(int) * (b_len + 1));
  hipMalloc(&d_max_scores, sizeof(int) * tn);
  hipMalloc(&d_a, sizeof(int) * a_len);
  hipMalloc(&d_b, sizeof(int) * b_len);
  hipMemcpy(d_a, a, sizeof(int) * a_len, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, sizeof(int) * b_len, hipMemcpyHostToDevice);
  hipMemset(d_scores, 0, sizeof(int) * (b_len+1));
  hipMemset(d_scores1, 0, sizeof(int) * (b_len+1));
  hipMemset(d_scores2, 0, sizeof(int) * (b_len+1));
  hipMemset(d_max_scores, 0, sizeof(int) * tn);

  for (int y = 0; y < a_len + b_len + 1; y++) {
    cuda_sw <<< blocks, threads >>>(d_a, d_b, a_len, b_len, d_scores, d_max_scores, y, d_scores1, d_scores2);
    int* t = d_scores2;
    d_scores2 = d_scores1;
    d_scores1 = d_scores;
    d_scores = t;
  }

  int *h_max_scores = new int[tn];
  hipMemcpy(h_max_scores, d_max_scores, sizeof(int) * tn, hipMemcpyDeviceToHost);

  for (int i = 0; i < tn; i++) {
    max_score = max(max_score, h_max_scores[i]);
  }

  return max_score;
}
